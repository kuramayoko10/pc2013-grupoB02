#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include "extern.h"
#include "smatrix.h"

#define THREADS_PER_BLOCK 250
#define check(X) 		\
{				\
	hipError_t cerr = X;		\
	if (cerr != hipSuccess){	\
		fprintf(stderr, "GPUassert:%s at line%d.\n", hipGetErrorString(cerr), __LINE__);	\
		abort();			\
	}					\
}
#define my_abs(x) (x)>=0.0?(x):-1.0*(x)

__global__ void kernel(float* A, float *b, float *x, float* oldx, float err, unsigned order) 
{ 
	unsigned idx = threadIdx.x + blockDim.x * blockIdx.x; 
	unsigned i;
	float aux;
	if(idx<order)
	{ 
		__shared__ float maxxdiff; 
		__shared__ float maxx; 
		maxxdiff=1.0; 
		maxx=1.0; 
		while((maxxdiff/maxx)>err)
		{ 
			x[idx] = 0.0; 
			maxxdiff = 0.0; 
			maxx=0.0; 
			for (i=0; i<order; ++i) 
			{ 
				x[idx]+=(i!=idx)?(smatrix_at(A, idx, i)*oldx[i]):0.0; 
			} 
			x[idx] = (1.0/smatrix_at(A, idx, idx))*(b[idx]-x[idx]); 
			aux = my_abs(x[idx]);
			if (aux > maxx)
				maxx = aux;
			aux = abs(x[idx]-oldx[idx]);
			if (aux > maxxdiff)
				maxxdiff = aux;
			oldx[idx]=x[idx]; 
		} 
	} 
}

float *solve(float *A, float *b)
{
	unsigned i;
	float *x;	
	float *gpu_A, *gpu_b, *gpu_x, *gpu_oldx;
	x = vector_new(order);
	for (i=0; i<order; ++i)
		x[i] = 1.0;
	check(hipMalloc((void **)&gpu_A, order * order * sizeof(float)));
	check(hipMalloc((void **)&gpu_b, order * sizeof(float)));
	check(hipMalloc((void **)&gpu_x, order * sizeof(float)));
	check(hipMalloc((void **)&gpu_oldx, order * sizeof(float)));
	check(hipMemcpy(gpu_A, A, order * order * sizeof(float), hipMemcpyHostToDevice));
	check(hipMemcpy(gpu_b, b, order * sizeof(float), hipMemcpyHostToDevice));
	check(hipMemcpy(gpu_x, x, order * sizeof(float), hipMemcpyHostToDevice));
	check(hipMemcpy(gpu_oldx, x, order * sizeof(float), hipMemcpyHostToDevice));
	kernel<<<order/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(gpu_A, gpu_b, gpu_x, gpu_oldx, err, order);
	check(hipMemcpy(x, gpu_x, order * sizeof(float), hipMemcpyDeviceToHost));
	hipFree(gpu_A);
	hipFree(gpu_b);
	hipFree(gpu_x);
	hipFree(gpu_oldx);
	vector_print(x);
	return x;
}

