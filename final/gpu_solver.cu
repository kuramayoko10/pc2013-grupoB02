#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include "extern.h"
#include "smatrix.h"

#define check(X) assert(X==hipSuccess)

__global__ void kernel(float *a, float *b, float *c, unsigned order)
{
	int i = threadIdx.x + blockIdx.x;
	if (i<order)
		c[i] = a[i]+b[i];
}



float *solve(float *A, float *b)
{
	unsigned i, it=0;
	float *x;	
	float *gpu_A, *gpu_x, *gpu_b, *gpu_c;
	x = vector_new(order);
	for (i=0; i<order; ++i)
		x[i] = 1.0;
	check(hipMalloc((void **)&gpu_A, order * order *sizeof(float)));
	check(hipMalloc((void **)&gpu_b, order * sizeof(float)));
	check(hipMemcpy(gpu_A, A, order * order * sizeof(float), hipMemcpyHostToDevice));
	check(hipMemcpy(gpu_b, b, order * sizeof(float), hipMemcpyHostToDevice));
	while (it<it_num)
	{
		hipMemcpy(gpu_x, x, order * sizeof(float), hipMemcpyHostToDevice);
		kernel<<<order, 1>>>(gpu_x, gpu_b, gpu_c, order);	
		hipMemcpy(x, gpu_x, order * sizeof(float), hipMemcpyDeviceToHost);
		++it;
	}
	hipFree(gpu_A);
	hipFree(gpu_b);
	vector_print(x);
	return x;
}

