#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <assert.h>
#include "extern.h"
#include "smatrix.h"

__global__ void kernel(float *a, float *b, float *c, unsigned order)
{
	int i = threadIdx.x + blockIdx.x;
	if (i<order)
		c[i] = a[i]+b[i];
}



float *solve(float *A, float *b)
{
	unsigned i;
	float *x;	
	float *gpu_x, *gpu_b, *gpu_c;
	x = vector_new(order);
	for (i=0; i<order; ++i)
		x[i] = 1;
	hipMalloc((void **)&gpu_x, order * sizeof(float));
	hipMalloc((void **)&gpu_b, order * sizeof(float));
	hipMalloc((void **)&gpu_c, order * sizeof(float));
	assert(hipMemcpy(gpu_x, x, order * sizeof(float), hipMemcpyHostToDevice)==hipSuccess);
	assert(hipMemcpy(gpu_b, b, order * sizeof(float), hipMemcpyHostToDevice)==hipSuccess);
	kernel<<<order, 1>>>(gpu_x, gpu_b, gpu_c, order);	
	assert(hipMemcpy(x, gpu_c, order * sizeof(float), hipMemcpyDeviceToHost)==hipSuccess);
	vector_print(x);
	hipFree(gpu_c);
	hipFree(gpu_b);
	hipFree(gpu_x);
	return x;
}
